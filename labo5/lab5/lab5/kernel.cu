#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <cmath>
# include <chrono>

using namespace std;
#define SIZE 2048
#define THREADS 1024
#define N 1

__global__ void max_reduce(int* inputData, int dataSize, int offset) {
    int idx = threadIdx.x;

    if (idx < dataSize) {
        for (int s = 1; s < dataSize; s *= 2) {
            if (idx < dataSize / (2 * s)) {

                int l = inputData[(idx * 2) + offset];
                int r = inputData[(idx * 2) + 1 + offset];
                inputData[idx + offset] = max(l, r);

            }
            __syncthreads();
        }
    }
    __syncthreads();
    
}
__global__ void min_reduce(int* inputData, int dataSize, int offset) {
    int idx = threadIdx.x;

    if (idx < dataSize) {
        for (int s = 1; s < dataSize; s *= 2) {
            if (idx < dataSize / (2 * s)) {

                int l = inputData[(idx * 2) + offset];
                int r = inputData[(idx * 2) + 1 + offset];
                inputData[idx + offset] = min(l, r);

            }
            __syncthreads();
        }
    }
    __syncthreads();

}

__global__ void sum_reduce(int* arr, int dataSize, int offset) {
    int idx = threadIdx.x;
    if (idx < dataSize) {
        for (int s = 1; s < dataSize; s *= 2) {
            if (idx < dataSize / (2 * s)) {
                int l = arr[(idx * 2) + offset];
                int r = arr[(idx * 2) + 1 + offset];
                arr[idx + offset] = l + r;
            }
            __syncthreads();
        }
    }
    __syncthreads();
}
__global__ void product_reduce(int* arr, int dataSize, int offset) {
    int idx = threadIdx.x;
    if (idx < dataSize) {
        for (int s = 1; s < dataSize; s *= 2) {
            if (idx < dataSize / (2 * s)) {
                int l = arr[(idx * 2) + offset];
                int r = arr[(idx * 2) + 1 + offset];
                arr[idx + offset] = l * r;
            }
            __syncthreads();
        }
    }
    __syncthreads();
}

__global__ void sum_kernel(int* arr, int* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < SIZE) {
        atomicAdd(result, arr[tid]);
    }
}

__global__ void product_kernel(int* arr, int* result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < SIZE) {
        *result *= arr[tid];
    }
}


void sync() {

    /*int* test = (int*)malloc(SIZE* 4 * sizeof(int));
    for (int j = 0; j < 4* N; j++) {
        test[j] = N - j;
    }*/
    const int threads_per_block = THREADS;
    int blocks_per_grid = (SIZE + threads_per_block - 1) / threads_per_block;
    //Sum
    //---------------------------------------------------------------------
    //const auto startSync = std::chrono::steady_clock::now();
    int* arr_sum = (int*)malloc(SIZE * sizeof(int));
    int* arr_sum_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_sum[i] = 1;
    }
   
    int* d_arr_sum = NULL;
    int* d_arr_sum_res = NULL;
    hipMalloc((void**)&d_arr_sum, SIZE * sizeof(int));
    hipMalloc((void**)&d_arr_sum_res, SIZE * sizeof(int));

    //Product
    //---------------------------------------------------------------------
    int* arr_product = (int*)malloc(SIZE * sizeof(int));
    int* arr_product_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_product[i] = 1;
    }

    int* d_arr_product = NULL;
    int* d_arr_product_res = NULL;
    hipMalloc((void**)&d_arr_product, SIZE * sizeof(int));
    hipMalloc((void**)&arr_product_res, SIZE * sizeof(int));

    
    //Min
    //---------------------------------------------------------------------
    int* arr_min = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_min[i] = 1;
    }

    int* d_arr_min = NULL;
    hipMalloc((void**)&d_arr_min, SIZE * sizeof(int));

    //Max
    //---------------------------------------------------------------------
    int* arr_max = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_max[i] = 1;
    }

    int* d_arr_max = NULL;
    hipMalloc((void**)&d_arr_max, SIZE * sizeof(int));

    /*const auto endSync = std::chrono::steady_clock::now();
    const std::chrono::duration<double> elapsed_seconds{ endSync - startSync };
    cout << "ArrayTime: " << elapsed_seconds.count() << "\n";*/

    //Sum GPU
    //---------------------------------------------------------------------
    hipMemcpy(d_arr_sum, arr_sum, SIZE * sizeof(int), hipMemcpyHostToDevice);
    sum_kernel <<<blocks_per_grid, threads_per_block >>> (d_arr_sum, d_arr_sum_res);
    hipMemcpy(arr_sum_res, d_arr_sum_res, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    //Product GPU
    //---------------------------------------------------------------------
    hipMemcpy(d_arr_product, arr_product, SIZE * sizeof(int), hipMemcpyHostToDevice);
    product_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_product, d_arr_product_res);
    hipMemcpy(arr_product_res, d_arr_product, SIZE * sizeof(int), hipMemcpyDeviceToHost);


    //Min GPU
    //---------------------------------------------------------------------
    hipMemcpy(d_arr_min, arr_min, SIZE * sizeof(int), hipMemcpyHostToDevice);
    min_reduce << <1, 1024 >> > (d_arr_min, 1024, 0);
    hipMemcpy(arr_min, d_arr_min, SIZE * sizeof(int), hipMemcpyDeviceToHost);


    //Max GPU
    //---------------------------------------------------------------------
    hipMemcpy(d_arr_max, arr_max, SIZE * sizeof(int), hipMemcpyHostToDevice);

    
    max_reduce << <1, 1024 >> > (d_arr_max, 1024, 0);
    hipMemcpy(arr_max, d_arr_max, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    //
    // cout << "test2 \n";

    
    //hipDeviceSynchronize();
    
    hipFree(d_arr_sum);
    hipFree(d_arr_product);
    hipFree(d_arr_min);
    hipFree(d_arr_max);
    free(arr_sum);
    //free(test);
    free(arr_product);
    free(arr_min);
    free(arr_max);
}
void sync_async_copy() {

    /*int* test = (int*)malloc(SIZE *4* sizeof(int));
    for (int j = 0; j < 4*N; j++) {
        test[j] = N - j;
    }*/

    const int threads_per_block = THREADS;
    int blocks_per_grid = (SIZE + threads_per_block - 1) / threads_per_block;
    //Sum
    //---------------------------------------------------------------------
    int* arr_sum = (int*)malloc(SIZE * sizeof(int));
    int* arr_sum_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_sum[i] = 1;
    }

    int* d_arr_sum = NULL;
    int* d_arr_sum_res = NULL;
    hipMalloc((void**)&d_arr_sum, SIZE * sizeof(int));
    hipMalloc((void**)&d_arr_sum_res, SIZE * sizeof(int));

    //Product
    //---------------------------------------------------------------------
    int* arr_product = (int*)malloc(SIZE * sizeof(int));
    int* arr_product_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_product[i] = 1;
    }

    int* d_arr_product = NULL;
    int* d_arr_product_res = NULL;
    hipMalloc((void**)&d_arr_product, SIZE * sizeof(int));
    hipMalloc((void**)&arr_product_res, SIZE * sizeof(int));


    //Min
    //---------------------------------------------------------------------
    int* arr_min = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_min[i] = 1;
    }

    int* d_arr_min = NULL;
    hipMalloc((void**)&d_arr_min, SIZE * sizeof(int));

    //Max
    //---------------------------------------------------------------------
    int* arr_max = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_max[i] = 1;
    }

    int* d_arr_max = NULL;
    hipMalloc((void**)&d_arr_max, SIZE * sizeof(int));

    //Sum GPU
    //---------------------------------------------------------------------
   /* hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);*/

    hipMemcpyAsync(d_arr_sum, arr_sum, SIZE * sizeof(int), hipMemcpyHostToDevice);
    sum_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_sum, d_arr_sum_res);
    hipMemcpyAsync(arr_sum_res, d_arr_sum_res, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    /*hipEventSynchronize(start);
    hipEventRecord(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    cout << "Cudatime: " << ms << "\n";*/
    hipDeviceSynchronize();
    //Product GPU
    //---------------------------------------------------------------------
    hipMemcpyAsync(d_arr_product, arr_product, SIZE * sizeof(int), hipMemcpyHostToDevice);
    product_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_product, d_arr_product_res);
    hipMemcpyAsync(arr_product_res, d_arr_product, SIZE * sizeof(int), hipMemcpyDeviceToHost);


    //Min GPU
    //---------------------------------------------------------------------
    hipMemcpyAsync(d_arr_min, arr_min, SIZE * sizeof(int), hipMemcpyHostToDevice);

    /* int offset = 0;

     for (int i = 0; i < ceil(fmax((SIZE / 2048), 1)); i++) {
         if (SIZE - 2048 * i < 2048) {
             min_reduce << <1, SIZE / 2 - 2048 * i >> > (d_arr_min, SIZE - 2048 * i, offset);
         }
         else {
             min_reduce << <1, 1024 >> > (d_arr_min, 2048, offset);
         }
         offset += 2048;
     }*/
    min_reduce << <1, THREADS >> > (d_arr_min, SIZE, 0);
    hipMemcpyAsync(arr_min, d_arr_min, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //Max GPU
    //---------------------------------------------------------------------
    hipMemcpy(d_arr_max, arr_max, SIZE * sizeof(int), hipMemcpyHostToDevice);

    /*offset = 0;

    for (int i = 0; i < ceil(fmax((SIZE / 2048), 1)); i++) {
        if (SIZE - 2048 * i < 2048) {
            max_reduce << <1, SIZE / 2 - 2048 * i >> > (d_arr_max, SIZE - 2048 * i, offset);
        }
        else {
            max_reduce << <1, 1024 >> > (d_arr_max, 2048, offset);
        }
        offset += 2048;
    }*/
    max_reduce << <1, THREADS >> > (d_arr_max, SIZE, 0);
    hipMemcpy(arr_max, d_arr_max, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(d_arr_sum);
    hipFree(d_arr_product);
    hipFree(d_arr_min);
    hipFree(d_arr_max);
    free(arr_sum);
    //free(test);
    free(arr_product);
    free(arr_min);
    free(arr_max);
}

void async_with_streams() {
    hipStream_t stream_sum, stream_product, stream_min, stream_max;
    hipStreamCreate(&stream_sum);
    hipStreamCreate(&stream_product);
    hipStreamCreate(&stream_min);
    hipStreamCreate(&stream_max);
    const int threads_per_block = THREADS;
    int blocks_per_grid = (SIZE + threads_per_block - 1) / threads_per_block;
    //Sum
    //---------------------------------------------------------------------
    int* arr_sum = (int*)malloc(SIZE * sizeof(int));
    int* arr_sum_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_sum[i] = 1;
    }
    /*int* test = (int*)malloc(SIZE * sizeof(int));
    for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    
    int* d_arr_sum = NULL;
    int* d_arr_sum_res = NULL;
    hipMalloc((void**)&d_arr_sum, SIZE * sizeof(int));
    hipMalloc((void**)&d_arr_sum_res, SIZE * sizeof(int));

    hipMemcpyAsync(d_arr_sum, arr_sum, SIZE * sizeof(int), hipMemcpyHostToDevice, stream_sum);
    sum_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_sum, d_arr_sum_res);
    hipMemcpyAsync(arr_sum_res, d_arr_sum_res, SIZE * sizeof(int), hipMemcpyDeviceToHost,stream_sum);
    
    //Product
    //---------------------------------------------------------------------
    int* arr_product = (int*)malloc(SIZE * sizeof(int));
    int* arr_product_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_product[i] = 1;
    }
    
    /*for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    int* d_arr_product = NULL;
    int* d_arr_product_res = NULL;
    
    hipMalloc((void**)&d_arr_product, SIZE * sizeof(int));
    hipMalloc((void**)&d_arr_product_res, SIZE * sizeof(int));
    hipMemcpyAsync(d_arr_product, arr_product, SIZE * sizeof(int), hipMemcpyHostToDevice, stream_product);
    
    product_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_product, d_arr_product_res);
    hipMemcpyAsync(arr_product_res, d_arr_product_res, SIZE * sizeof(int), hipMemcpyDeviceToHost, stream_product);
    
    //Min
    //---------------------------------------------------------------------
    int* arr_min = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_min[i] = 100;
    }
  
    /*for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    int* d_arr_min = NULL;

    hipMalloc((void**)&d_arr_min, SIZE * sizeof(int));
    hipMemcpyAsync(d_arr_min, arr_min, SIZE * sizeof(int), hipMemcpyHostToDevice, stream_min);

    min_reduce << <1, THREADS >> > (d_arr_min, SIZE, 0);
    hipMemcpyAsync(arr_min, d_arr_min, SIZE * sizeof(int), hipMemcpyDeviceToHost,stream_min);

    //Max
    //---------------------------------------------------------------------
    int* arr_max = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_max[i] = 1;
    }
   
    /*for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    int* d_arr_max = NULL;
   
    hipMalloc((void**)&d_arr_max, SIZE * sizeof(int));
    hipMemcpyAsync(d_arr_max, arr_max, SIZE * sizeof(int), hipMemcpyHostToDevice,stream_max);

    
    max_reduce << <1, THREADS >> > (d_arr_max, SIZE, 0);
    hipMemcpyAsync(arr_max, d_arr_max, SIZE * sizeof(int), hipMemcpyDeviceToHost,stream_max);


    hipStreamSynchronize(stream_sum);
    hipStreamSynchronize(stream_product);
    hipStreamSynchronize(stream_min);
    hipStreamSynchronize(stream_max);
    hipStreamDestroy(stream_sum);
    hipStreamDestroy(stream_product);
    hipStreamDestroy(stream_min);
    hipStreamDestroy(stream_max);

    hipDeviceSynchronize();

    hipFree(d_arr_sum);
    hipFree(d_arr_product);
    hipFree(d_arr_min);
    hipFree(d_arr_max);
    free(arr_sum);
    //free(test);
    free(arr_product);
    free(arr_min);
    free(arr_max);
}

void async_without_streams() {
    const int threads_per_block = THREADS;
    int blocks_per_grid = (SIZE + threads_per_block - 1) / threads_per_block;
    //Sum
    //---------------------------------------------------------------------
    int* arr_sum = (int*)malloc(SIZE * sizeof(int));
    int* arr_sum_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_sum[i] = 1;
    }
    /*int* test = (int*)malloc(SIZE * sizeof(int));
    for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    int* d_arr_sum = NULL;
    int* d_arr_sum_res = NULL;
    hipMalloc((void**)&d_arr_sum, SIZE * sizeof(int));
    hipMalloc((void**)&d_arr_sum_res, SIZE * sizeof(int));

    hipMemcpyAsync(d_arr_sum, arr_sum, SIZE * sizeof(int), hipMemcpyHostToDevice);
    sum_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_sum, d_arr_sum_res);
    hipMemcpyAsync(arr_sum_res, d_arr_sum_res, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    //Product
    //---------------------------------------------------------------------
    int* arr_product = (int*)malloc(SIZE * sizeof(int));
    int* arr_product_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_product[i] = 1;
    }
    int* d_arr_product = NULL;
    int* d_arr_product_res = NULL;
   
    /*for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    hipMalloc((void**)&d_arr_product, SIZE * sizeof(int));
    hipMalloc((void**)&d_arr_product_res, SIZE * sizeof(int));
    hipMemcpyAsync(d_arr_product, arr_product, SIZE * sizeof(int), hipMemcpyHostToDevice);

    product_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_product, d_arr_product_res);
    hipMemcpyAsync(arr_product_res, d_arr_product_res, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    //Min
    //---------------------------------------------------------------------
    int* arr_min = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_min[i] = 100;
    }

    int* d_arr_min = NULL;
    /*for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    hipMalloc((void**)&d_arr_min, SIZE * sizeof(int));
    hipMemcpyAsync(d_arr_min, arr_min, SIZE * sizeof(int), hipMemcpyHostToDevice);

   /* int offset = 0;

    for (int i = 0; i < ceil(fmax((SIZE / 2048), 1)); i++) {
        if (SIZE - 2048 * i < 2048) {
            min_reduce << <1, SIZE / 2 - 2048 * i >> > (d_arr_min, SIZE - 2048 * i, offset);
        }
        else {
            min_reduce << <1, 1024 >> > (d_arr_min, 2048, offset);
        }
        offset += 2048;
    }*/
    min_reduce << <1, THREADS >> > (d_arr_min, SIZE, 0);
    hipMemcpyAsync(arr_min, d_arr_min, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    //Max
    //---------------------------------------------------------------------
    int* arr_max = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_max[i] = 1;
    }

    int* d_arr_max = NULL;
    /*for (int j = 0; j < N; j++) {
        test[j] = N - j;
    }*/
    hipMalloc((void**)&d_arr_max, SIZE * sizeof(int));
    hipMemcpyAsync(d_arr_max, arr_max, SIZE * sizeof(int), hipMemcpyHostToDevice);

    /*offset = 0;

    for (int i = 0; i < ceil(fmax((SIZE / 2048), 1)); i++) {
        if (SIZE - 2048 * i < 2048) {
            max_reduce << <1, SIZE / 2 - 2048 * i >> > (d_arr_max, SIZE - 2048 * i, offset);
        }
        else {
            max_reduce << <1, 1024 >> > (d_arr_max, 2048, offset);
        }
        offset += 2048;
    }*/
    max_reduce << <1, THREADS >> > (d_arr_max, SIZE, 0);
    hipMemcpyAsync(arr_max, d_arr_max, SIZE * sizeof(int), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    //free(test);
    hipFree(d_arr_sum);
    hipFree(d_arr_product);
    hipFree(d_arr_min);
    hipFree(d_arr_max);
    free(arr_sum);

  
    free(arr_product);
    free(arr_min);
    free(arr_max);
}

float kernel_timings() {
    const int threads_per_block = THREADS;
    int blocks_per_grid = (SIZE + threads_per_block - 1) / threads_per_block;
    //Sum
    //---------------------------------------------------------------------


    const auto startSync = std::chrono::steady_clock::now();
  
    int* arr_sum = (int*)malloc(SIZE * sizeof(int));
    int* arr_sum_res = (int*)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i) {
        arr_sum[i] = 1;
    }
    const auto endSync = std::chrono::steady_clock::now();
    const std::chrono::duration<double> elapsed_seconds{ endSync - startSync };
    float ms2 = elapsed_seconds.count();
    cout << "Execution time sync: " << ms2 << "\n";

    int* d_arr_sum = NULL;
    int* d_arr_sum_res = NULL;
    hipMalloc((void**)&d_arr_sum, SIZE * sizeof(int));
    hipMalloc((void**)&d_arr_sum_res, SIZE * sizeof(int));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    

    hipMemcpyAsync(d_arr_sum, arr_sum, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(start);
    //product_kernel << <blocks_per_grid, threads_per_block >> > (d_arr_sum, d_arr_sum_res);
    max_reduce << <1, THREADS >> > (d_arr_sum, SIZE, 0);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    //std::cout << "Kernel execution time: " << ms << " milliseconds" << std::endl;
    std::cout << ms << std::endl;
    hipMemcpyAsync(arr_sum_res, d_arr_sum_res, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    free(arr_sum);
    free(arr_sum_res);
    hipFree(d_arr_sum);
    hipFree(d_arr_sum_res);
    return ms2;
}



int main()
{
    /*int n = 1000;
    float total = 0;
    for (int i = 0; i < n; i++) {
        total = kernel_timings();
    }
    float avg = total / n;
    cout << "avg: " << avg;*/
    float* s = (float*)malloc(101 * sizeof(float));
    float* s1 = (float*)malloc(101 * sizeof(float));
    float* s2 = (float*)malloc(101 * sizeof(float));
    float* s3 = (float*)malloc(101 * sizeof(float));
    for (int j = 0; j < 101; j++) {


        int n = N;
        double totalSync = 0;
        double totalSyncA = 0;
        double totalASyncS = 0;
        double totalASyncNS = 0;
        for (int i = 0; i < n; i++) {
            //cout << "Iteration " << i << "\n";
            const auto startSync = std::chrono::steady_clock::now();
            sync();
            const auto endSync = std::chrono::steady_clock::now();
            const std::chrono::duration<double> elapsed_seconds{ endSync - startSync };
            //cout << "Execution time sync: " << elapsed_seconds.count() << "\n";
            totalSync += elapsed_seconds.count();

            const auto startSync2 = std::chrono::steady_clock::now();
            sync_async_copy();
            const auto endSync2 = std::chrono::steady_clock::now();
            const std::chrono::duration<double> elapsed_seconds4{ endSync2 - startSync2 };
            //cout << "Execution time sync: " << elapsed_seconds4.count() << "\n";
            totalSyncA += elapsed_seconds4.count();

            const auto startAsync = std::chrono::steady_clock::now();
            async_with_streams();
            const auto endAsync = std::chrono::steady_clock::now();
            const std::chrono::duration<double> elapsed_seconds2{ endAsync - startAsync };
            //cout << "Execution time async with streams: " << elapsed_seconds2.count() << "\n";
            totalASyncS += elapsed_seconds2.count();

            const auto startAsync2 = std::chrono::steady_clock::now();
            async_without_streams();
            const auto endAsync2 = std::chrono::steady_clock::now();
            const std::chrono::duration<double> elapsed_seconds3{ endAsync2 - startAsync2 };
            //cout << "Execution time async without streams: " << elapsed_seconds3.count() << "\n";
            totalASyncNS += elapsed_seconds3.count();
        }

        /*cout << "Sync: " << (totalSync / n) << "\n";
        cout << "SyncA: " << (totalSyncA / n) << "\n";
        cout << "SyncAS: " << (totalASyncS / n) << "\n";
        cout << "SyncANS: " << (totalASyncNS / n) << "\n";
        cout << "\n";*/
        /*cout  << (totalSync / n) << "\n";
        cout << (totalSyncA / n) << "\n";
        cout << (totalASyncS / n) << "\n";
        cout  << (totalASyncNS / n) << "\n";
        cout << "\n";*/
        s[j] = (totalSync / n);
        s1[j] = (totalSyncA / n);
        s2[j] = (totalASyncS / n);
        s3[j] = (totalASyncNS / n);
    }
    for (int i = 0; i < 101; i++) {
        cout << s[i] << "\n";
    }
    cout << "\n";
    for (int i = 0; i < 101; i++) {
        cout << s1[i] << "\n";
    }
    cout << "\n";
    for (int i = 0; i < 101; i++) {
        cout << s2[i] << "\n";
    }
    cout << "\n";
    for (int i = 0; i < 101; i++) {
        cout << s3[i] << "\n";
    }
    return 0;
}