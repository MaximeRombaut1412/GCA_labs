#include "hip/hip_runtime.h"
﻿#include <ctime>
#include <fstream>
#include <iostream>  
#include <sstream>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iomanip>
#include <chrono>
using namespace std;
//https://reasonabledeviations.com/2019/10/02/k-means-in-cpp/#c-preambles

struct Location {
    float lat, lon;
    int cluster;
    double minDistance;

    //Constructors
    Location() : 
        lat(0.0),
        lon(0.0),
        cluster(-1),
        minDistance(DBL_MAX) {}

    Location(float lat, float lon) :
        lat(lat),
        lon(lon),
        cluster(-1),
        minDistance(DBL_MAX) {}
    Location(float lat, float lon, int cluster) :
        lat(lat),
        lon(lon),
        cluster(cluster),
        minDistance(DBL_MAX) {}
};

__global__ void AssignLocationKernel(Location* locations, Location* centers, int amount_of_locations, int amount_of_centers) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < amount_of_locations) {
		for (int j = 0; j < amount_of_centers; j++) {
			double distance = sqrt((locations[i].lat - centers[j].lat) * (locations[i].lat - centers[j].lat) + (locations[i].lon - centers[j].lon) * (locations[i].lon - centers[j].lon));
			if (distance < locations[i].minDistance) {
				locations[i].minDistance = distance;
				locations[i].cluster = j;
			}
		}
	}
}

//struct Center {
//    float lat, lon;
//    int cluster_number;
//    Center(float lat, float lon, int cluster_number) :
//        lat(lat),
//        lon(lon),
//        cluster_number(cluster_number){}
//};

std::ostream& operator<<(std::ostream& os, const Location& location) {
    os << "(" << location.lat << ", " << location.lon << ")";
    return os;
}

double calculateDistanceCPU(Location a, Location b) {
    return sqrt((a.lat - b.lat) * (a.lat - b.lat) + (a.lon - b.lon) * (a.lon - b.lon));
}

//Read locations out of csv file and put them in vector
vector<Location> readLocationsCsv() {
    vector<Location> locations;
    string line;
    ifstream inputfile("locations_50000.csv");
    while (getline(inputfile, line)) {
        stringstream lineStream(line);
        string bit;
        float lat, lon;
        getline(lineStream, bit, ',');
        lat = stof(bit);
        getline(lineStream, bit, '\n');
        lon = stof(bit);
        locations.push_back(Location(lat, lon));
    }
    return locations;
}
void AssignLocationToCenter(Location* l, vector<Location>* centers) {
    for (int i = 0; i <centers->size(); i++) {
        double distance = calculateDistanceCPU(*l, centers->at(i));
        if (distance < l->minDistance) {
            l->minDistance = distance;
            l->cluster = i;
        }
    }
}

void AssignLocationToCenterGPU(vector<Location>* locations, vector<Location>* centers) {
    //Allocate memory on GPU
    Location* GPULocations = NULL;
    Location* GPUCenters = NULL;
    hipMalloc((void**)&GPULocations, locations->size()  *sizeof(Location));
    hipMalloc((void**)&GPUCenters, centers->size() * sizeof(Location));

    //Copy data from host to device
    hipMemcpy(GPULocations, locations->data(), locations->size() * sizeof(Location), hipMemcpyHostToDevice);
    hipMemcpy(GPUCenters, centers->data(), centers->size() * sizeof(Location), hipMemcpyHostToDevice);
    
    //Threads per block
    int threadsPerBlock = 1024;
    int blocksPerGrid = (locations->size() + threadsPerBlock - 1) / threadsPerBlock; //Can we optimize this?

    //Call kernel
    AssignLocationKernel<<<blocksPerGrid, threadsPerBlock>>>(GPULocations, GPUCenters, locations->size(), centers->size());

    //Copy data from device to host
    hipMemcpy(locations->data(), GPULocations, locations->size() * sizeof(Location), hipMemcpyDeviceToHost);

    //Free memory
    hipFree(GPULocations);
    hipFree(GPUCenters);

}
void WriteCentersToFile(vector<Location>* centers, int k) {
	std::ofstream myfile;
	myfile.open("output.csv", std::ios::app);
	myfile << "============================" << endl;
	myfile << "K: " << k << endl;
	myfile << "----------------------------" << endl;
	for (vector<Location>::iterator it = centers->begin();
		it != centers->end(); ++it) {
		myfile << "Cluster: " << it->cluster <<", latitude:  " << std::fixed << std::setprecision(6) << it->lat << ", longitude: " << std::fixed << std::setprecision(6) <<it->lon << endl;
	}
	myfile << "----------------------------" << endl;
	myfile.close();
}
void WriteCentersToFileGPU(vector<Location>* centers, int k) {
    std::ofstream myfile;
    myfile.open("outputGPU.csv", std::ios::app);
    myfile << "============================" << endl;
    myfile << "K: " << k << endl;
    myfile << "----------------------------" << endl;
    for (vector<Location>::iterator it = centers->begin();
        it != centers->end(); ++it) {
        myfile << "Cluster: " << it->cluster << ", latitude:  " << std::fixed << std::setprecision(6) << it->lat << ", longitude: " << std::fixed << std::setprecision(6) << it->lon << endl;
    }
    myfile << "----------------------------" << endl;
    myfile.close();
}


void resetOutputFiles() {
    std::ofstream outputFile;
    outputFile.open("output.csv", std::ios::trunc); // Open file in truncation mode
    outputFile.close(); // Close the file
    std::cout << "Cpu file reset successfully." << std::endl;
    std::ofstream outputFileGPU;
    outputFileGPU.open("outputGPU.csv", std::ios::trunc); // Open file in truncation mode
    outputFileGPU.close(); // Close the file
    std::cout << "Gpu file reset successfully." << std::endl;
}

bool CalculateCenterSums(vector<Location>* locations, vector<Location>* centers) {
    vector<float> lat_and_lon_sums; //Vector to add sums to. First element is the lat of the first center. The second element the lon of first. Third lat of second center,...
    vector<int> amount_of_locations; //The amount of locations per cluster. Needed to divide at the end
    for (int i = 0; i < centers->size(); i++) {
        lat_and_lon_sums.push_back(0);
        lat_and_lon_sums.push_back(0);
        amount_of_locations.push_back(0);
    }
    for (int i = 0; i < locations->size(); i++) {
        Location l = locations->at(i);
        amount_of_locations.at(l.cluster) += 1;
        int center = l.cluster *2;
        lat_and_lon_sums.at(center) += l.lat;
        lat_and_lon_sums.at(center + 1) += l.lon;
    }
    int clusters_with_same_values = 0;
    for (int i = 0; i < centers->size(); i++) {
        float lat = lat_and_lon_sums.at(i*2) / amount_of_locations.at(i);
        float lon = lat_and_lon_sums.at(i*2 + 1) / amount_of_locations.at(i);
        if(lat == centers->at(i).lat && lon == centers->at(i).lon){
            clusters_with_same_values++;
		}
        (*centers)[i].lat = lat;
        (*centers)[i].lon = lon;
    }
    if (clusters_with_same_values == centers->size()) {
		return true;
	}
    
    return false;
}
//Here we calculate the clusters with the CPU.
//Paramters:
//  * locations: locations that need to be clustered
//  * iterations: amount of iterations before quiting
//  * k: amount of clusters that will be used
void KmeansCPU(vector<Location>* locations, int iterations, int k, vector<Location>* centers) {
    /*vector<Location> centers;
    int amount_of_locations = locations->size();
    srand(time(0));
    for (int i = 0; i < k; i++) {
        Location l = locations->at(rand() % amount_of_locations);
        centers.push_back(Location(l.lat,l.lon, i));
    }*/
    int amount_of_locations = locations->size();
    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < amount_of_locations; i++) {
            AssignLocationToCenter(&locations->at(i), centers);
        }
        if (CalculateCenterSums(locations, centers)) {
            WriteCentersToFile(centers,k);
            break;
        }
        /*else {
            WriteCentersToFile(&centers);
        }*/
    }
}

void KmeansGPU(vector<Location>* locations, int iterations, int k, vector<Location>* centers) {
	/*vector<Location> centers;
	int amount_of_locations = locations->size();
	srand(time(0));
	for (int i = 0; i < k; i++) {
		Location l = locations->at(rand() % amount_of_locations);
		centers.push_back(Location(l.lat, l.lon, i));
	}*/


	for (int j = 0; j < iterations; j++) {
        AssignLocationToCenterGPU(locations, centers);
		if (CalculateCenterSums(locations, centers)) {
			WriteCentersToFileGPU(centers, k);
			break;
		}
		/*else {
			WriteCentersToFile(&centers);
		}*/
	}
}



void ResetLocationClusters(vector<Location>* locations) {
	for (int i = 0; i < locations->size(); i++) {
		locations->at(i).cluster = -1;
		locations->at(i).minDistance = DBL_MAX;
	}
}

vector<Location> GetCenterLocations(vector<Location>* locations, int k) {
    vector<Location> centers;
    int amount_of_locations = locations->size();
    srand(time(0));
    for (int i = 0; i < k; i++) {
        Location l = locations->at(rand() % amount_of_locations);
        centers.push_back(Location(l.lat, l.lon, i));
    }
    return centers;
}


int main()
{

    resetOutputFiles();
    vector<Location> locations= readLocationsCsv(); //Get locations
    int k = 24;
    vector<Location> centers = GetCenterLocations(&locations, k);
    for (int i = 0; i < k/2; i++) {
        //Execute Kmeans on CPU
        const auto startCPU = std::chrono::steady_clock::now();
        KmeansCPU(&locations, 1000, k-i*2, &centers);
        const auto endCPU = std::chrono::steady_clock::now();
        //Execute Kmeans on GPU
        const auto startGPU = std::chrono::steady_clock::now();
        KmeansGPU(&locations, 1000, k - i * 2, &centers);
        const auto endGPU = std::chrono::steady_clock::now();
        //Reset clusters
        ResetLocationClusters(&locations);
        const std::chrono::duration<double, milli> elapsed_secondsCPU{ endCPU - startCPU };
        const std::chrono::duration<double, milli> elapsed_secondsGPU{ endGPU - startGPU };
        cout << "CPU: " << elapsed_secondsCPU.count() << "ms" << endl;
        cout << "GPU: " << elapsed_secondsGPU.count() << "ms" << endl;
        //cout << "K: " << k - i * 2 << endl;
    }
    //KmeansCPU(&locations, 20, 5);
    cout << "test";
    return 0;
}

//NOTES
//--------------
//Indien veel clusters zouden we de max parallel kunnen bereken?
//Kijken of het berekenen van een nieuw centerpunt in parallel kan -> De som van lat en lon waarden van alle clusters moeten opgeteld worden en dan gedeeld om zo een average te krijgen.
//Mogelijk om de waarden visueel voor te stellen in een grafiek
//Enkel hiërarchieën weergeven
//
