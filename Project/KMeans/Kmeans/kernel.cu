#include "hip/hip_runtime.h"
﻿#include <ctime>
#include <fstream>
#include <iostream>  
#include <sstream>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iomanip>
using namespace std;
//https://reasonabledeviations.com/2019/10/02/k-means-in-cpp/#c-preambles

struct Location {
    float lat, lon;
    int cluster;
    double minDistance;

    //Constructors
    Location() : 
        lat(0.0),
        lon(0.0),
        cluster(-1),
        minDistance(DBL_MAX) {}

    Location(float lat, float lon) :
        lat(lat),
        lon(lon),
        cluster(-1),
        minDistance(DBL_MAX) {}
    Location(float lat, float lon, int cluster) :
        lat(lat),
        lon(lon),
        cluster(cluster),
        minDistance(DBL_MAX) {}
};

//struct Center {
//    float lat, lon;
//    int cluster_number;
//    Center(float lat, float lon, int cluster_number) :
//        lat(lat),
//        lon(lon),
//        cluster_number(cluster_number){}
//};

std::ostream& operator<<(std::ostream& os, const Location& location) {
    os << "(" << location.lat << ", " << location.lon << ")";
    return os;
}

double calculateDistanceCPU(Location a, Location b) {
    return sqrt((a.lat - b.lat) * (a.lat - b.lat) + (a.lon - b.lon) * (a.lon - b.lon));
}

//Read locations out of csv file and put them in vector
vector<Location> readLocationsCsv() {
    vector<Location> locations;
    string line;
    ifstream inputfile("locations_50000.csv");
    while (getline(inputfile, line)) {
        stringstream lineStream(line);
        string bit;
        float lat, lon;
        getline(lineStream, bit, ',');
        lat = stof(bit);
        getline(lineStream, bit, '\n');
        lon = stof(bit);
        locations.push_back(Location(lat, lon));
    }
    return locations;
}
void AssignLocationToCenter(Location* l, vector<Location>* centers) { //Fout??
    for (int i = 0; i <centers->size(); i++) {
        double distance = calculateDistanceCPU(*l, centers->at(i));
        if (distance < l->minDistance) {
            l->minDistance = distance;
            l->cluster = i;
        }
    }
}
void WriteCentersToFile(vector<Location>* centers, int k) {
	std::ofstream myfile;
	myfile.open("output.csv", std::ios::app);
	myfile << "============================" << endl;
	myfile << "K: " << k << endl;
	myfile << "----------------------------" << endl;
	for (vector<Location>::iterator it = centers->begin();
		it != centers->end(); ++it) {
		myfile << "Cluster: " << it->cluster <<", latitude:  " << std::fixed << std::setprecision(6) << it->lat << ", longitude: " << std::fixed << std::setprecision(6) <<it->lon << endl;
	}
	myfile << "----------------------------" << endl;
	myfile.close();
}


void resetOutputFile() {
    std::ofstream outputFile;
    outputFile.open("output.csv", std::ios::trunc); // Open file in truncation mode
    outputFile.close(); // Close the file
    std::cout << "File reset successfully." << std::endl;
}

bool CalculateCenterSums(vector<Location>* locations, vector<Location>* centers) {
    vector<float> lat_and_lon_sums; //Vector to add sums to. First element is the lat of the first center. The second element the lon of first. Third lat of second center,...
    vector<int> amount_of_locations; //The amount of locations per cluster. Needed to divide at the end
    for (int i = 0; i < centers->size(); i++) {
        lat_and_lon_sums.push_back(0);
        lat_and_lon_sums.push_back(0);
        amount_of_locations.push_back(0);
    }
    for (int i = 0; i < locations->size(); i++) {
        Location l = locations->at(i);
        amount_of_locations.at(l.cluster) += 1;
        int center = l.cluster *2;
        lat_and_lon_sums.at(center) += l.lat;
        lat_and_lon_sums.at(center + 1) += l.lon;
    }
    int clusters_with_same_values = 0;
    for (int i = 0; i < centers->size(); i++) {
        float lat = lat_and_lon_sums.at(i*2) / amount_of_locations.at(i);
        float lon = lat_and_lon_sums.at(i*2 + 1) / amount_of_locations.at(i);
        if(lat == centers->at(i).lat && lon == centers->at(i).lon){
            clusters_with_same_values++;
		}
        (*centers)[i].lat = lat;
        (*centers)[i].lon = lon;
    }
    if (clusters_with_same_values == centers->size()) {
		return true;
	}
    
    return false;
}
//Here we calculate the clusters with the CPU.
//Paramters:
//  * locations: locations that need to be clustered
//  * iterations: amount of iterations before quiting
//  * k: amount of clusters that will be used
void KmeansCPU(vector<Location>* locations, int iterations, int k) {
    vector<Location> centers;
    int amount_of_locations = locations->size();
    srand(time(0));
    for (int i = 0; i < k; i++) {
        Location l = locations->at(rand() % amount_of_locations);
        centers.push_back(Location(l.lat,l.lon, i));
    }

    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < amount_of_locations; i++) {
            AssignLocationToCenter(&locations->at(i), &centers);
        }
        if (CalculateCenterSums(locations, &centers)) {
            WriteCentersToFile(&centers,k);
            break;
        }
        /*else {
            WriteCentersToFile(&centers);
        }*/
    }
}
void ResetLocationClusters(vector<Location>* locations) {
	for (int i = 0; i < locations->size(); i++) {
		locations->at(i).cluster = -1;
		locations->at(i).minDistance = DBL_MAX;
	}
}


int main()
{

    resetOutputFile();
    vector<Location> locations= readLocationsCsv(); //Get locations
    int k = 24;
    for (int i = 0; i < k; i++) {
        KmeansCPU(&locations, 1000, k-i*2);
        ResetLocationClusters(&locations);
    }
    //KmeansCPU(&locations, 20, 5);
    cout << "test";
    return 0;
}

//NOTES
//--------------
//Indien veel clusters zouden we de max parallel kunnen bereken?
//Kijken of het berekenen van een nieuw centerpunt in parallel kan -> De som van lat en lon waarden van alle clusters moeten opgeteld worden en dan gedeeld om zo een average te krijgen.
//Mogelijk om de waarden visueel voor te stellen in een grafiek
//Enkel hiërarchieën weergeven
//
